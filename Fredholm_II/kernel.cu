#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>
#include <assert.h>
#include "hipblas.h"
#include "hipsolver.h"


#define a 0.0
#define b 1.0
#define n 100
//double K(double x, double y)
//{
//	return x * y;
//}
//
//__device__  double f(double x)
//{
//	return 2 * x;
//}
//
//__device__ double pi(double x)
//{
//	return 4.0 / (1.0 + x * x);
//}
//
//__device__ double phi(double x, int i)
//{
//	return __powf(x, i);
//}

double ut(double x)
{
	return 3 * x;
}



__device__ void hello()
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	printf("hello from %d\n", idx);
}


__global__ void Solve(double *dA, double *dF, double *dX0, double *dX1, int N)
{
	double aa, sum = 0;
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	if (t < N)
	{
		for (int j = 0; j < N; j++)
		{
			sum += dA[j + t * N] * dX0[j];
			//if (t == 0 && j == 0) printf("dA[%d]= %f\tdX0[%d]= %f\n", j + t * N, dA[j + t * N], j, dX0[j]);
			if (j == t)
				aa = dA[j + t * N];
		}
		dX1[t] = dX0[t] + (dF[t] - sum) / aa;
		printf("%f %f %f %f %f\n", dX1[t], dX0[t], dF[t], sum, aa);
	}
	
}

__global__ void _resolution(double *x, double *c, double *y, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		double sum = 0;
		for (int i = 0; i < size; i++)
		{
			sum += c[i] * pow(x[idx], i);
			//printf("%f   %f\n", c[i], x[idx]);
		}

		y[idx] = 3 * x[idx] + c[idx];
		//printf("\n\n%f   ", y[idx]);
	}
}


__global__ void KernelJacobi(double* deviceA, double* deviceF, double* deviceX0, double* deviceX1, int N)

{
	double temp;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		deviceX1[i] = deviceF[i];
		for (int j = 0; j < N; j++)
		{
			if (i != j)
				deviceX1[i] -= deviceA[j + i * N] * deviceX0[j];
			else
				temp = deviceA[j + i * N];
		}
		deviceX1[i] /= temp;
	}
}

//Raschetdeltidlyausloviaostanovki

__global__ void EpsJacobi(double* deviceX0, double* deviceX1, double* delta, int N)

{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		delta[i] += abs(deviceX0[i] - deviceX1[i]);
		deviceX0[i] = deviceX1[i];
	}
}

__global__ void kernel(double *A, double *f, double par, double *x0, double *x1, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		int ia = n * idx;
		double sum = 0;
		for (int i = 0; i < size; i++)
			sum += A[ia + i] * x0[i];
		x1[idx] = x0[idx] + par * (sum - f[idx]);
		//printf("%f\n", x1[idx]);
	}
}

__global__ void Eps(double *dX0, double *dX1, double *delta, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		//if (i == 0) printf("d0[%d]= %f\tdX1[%d]= %f\n", i, dX0[i], i, dX1[i]);
		delta[i] = abs(dX0[i] - dX1[i]);
		dX0[i] = dX1[i];
	}
}


__global__ void create_F(double *arr, double *f, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		double ff = 0;
		for (int i = 0; i < size; i++)
			ff += pow(arr[i], idx + 2);
		ff *= 2;
		ff /= size;
		f[idx] = ff;
	}
}

__global__ void createMatrix(double *A, double *arr, int size)
{
	int ind = size * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	//printf("index = %d", ind);
	if (ind < size * size)
	{
		double alpha = 0;
		double beta = 0;
		for (int i = 0; i < size; i++)
		{
			alpha += __powf(arr[i], blockDim.y * blockIdx.y + threadIdx.y + blockDim.x * blockIdx.x + threadIdx.x + 2);
			beta += (__powf(arr[i], blockDim.x * blockIdx.x + threadIdx.x + 2) / (blockDim.y * blockIdx.y + threadIdx.y + 3));
		/*	if (i == 0 && ind == 0)
				printf("alpha=%f\tbeta=%f\n", alpha, beta);*/
		}

		alpha /= size;
		beta /= size;
		A[ind] = alpha - beta;
		//sum = 0;
	}
	//A[threadIdx.y*size + threadIdx.x] = integral(arr, size, sum);//10 * threadIdx.y + threadIdx.x;
	//printf("A[%d]= %f\n", A[threadIdx.y*size + threadIdx.x]);
}




__global__ void resolution(double *x, double *c, double *y, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		double sum = 0;
		for (int i = 0; i < size; i++)
			sum += c[i] * pow(x[idx], i);
		y[idx] = sum;
	}
}

int main()
{
	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	int m = n;
	const int lda = m;
	const int ldb = m;
	const int nrhs = 1; // number of right hand side vectors
	size_t size = m * m * sizeof(double);
	//double h = (b - a) / (n - 1); //���
	//double sum_h = 0;
	//double EPS = 1.e-5;
	double *h_A = new double[size]; // �������
	double *x_h = new double[n]; //�����
	double *hx_int = new double[n]; //����� ��� ���������
	double *f_h = new double[n]; //������ �����
	//double *h_x0 = new double[n]; //����������� x(n)
	//double *h_x1 = new double[n]; //����������� x(n+1)
	//double *h_delta = new double[n]; //������� |x(n+1)-x(n)|
	double *xc = new double[n];
	double *dx_int, *d_A, *f_d, *d_tau, *d_work, *x_d;
	int lwork = 0, *devInfo;
	int info_gpu = 0;
	const double one = 1;


	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	cudaStat2 = hipMalloc((void**)&d_tau, sizeof(double)*n);

	hipEvent_t start, stop;
	double time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int N_threads = 32;
	int blockSize = N_threads * N_threads;
	int blocks = n / blockSize + (n % blockSize == 0 ? 0 : 1);
	hipMalloc((void **)&x_d, sizeof(double)*n);


	hipMalloc((void **)&dx_int, sizeof(double)*n); //����� ��� ���������
	//hipMalloc((void **)&d_x0, sizeof(double)*n); //����������� x(n)
	//hipMalloc((void **)&d_x1, sizeof(double)*n); //����������� x(n+1)
	//hipMalloc((void **)&d_delta, sizeof(double)*n);//������� |x(n+1)-x(n)|



	cudaStat1 = hipMalloc((void **)&d_A, size); //�������
	cudaStat2 = hipMalloc((void **)&d_tau, sizeof(double)*n);
	cudaStat3 = hipMalloc((void **)&f_d, sizeof(double)*n); //������ �����
	cudaStat4 = hipMalloc((void **)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);
	//hipMemcpy(sum_d, &sum_h, sizeof(double), hipMemcpyHostToDevice);
	int i = 0, j = 0;
	dim3 threadsPerBlock = dim3(N_threads, N_threads);
	dim3 blocksPerGrid = dim3(blocks * 5, blocks * 5);
	/*for (double x = a + i*h; x <= b; x += h)
	{
		x_h[i] = x;
		i++;
	}
	i = 0;*/
	double h = (b - a) / n; //���
	for (double x = a + 0.5 * h; x < b; x += h)
	{
		hx_int[i] = x;
		i++;
	}
	//printf("x[%d]= %f  x[%d]= %f\n", 0, x_h[0], n - 1, x_h[n - 1]);
	/*for (i = 0; i < n; i++)
	{
		printf("x[%d]= %f\n ", i, x_h[i]);
	}
	printf("\n");*/
	//for (i = 0; i < n; i++)
	//	h_x0[i] = 1;
	/*for (int i = 0; i < m; i++)
		for (int j = 0; j < m; j++)
			h_A[j * m + i] = 0;*/
	/*double eps = 1;
	int k = 0;*/
	//hipMemcpy(x_d, x_h, sizeof(double)*n, hipMemcpyHostToDevice); //�����
	hipMemcpy(dx_int, hx_int, sizeof(double)*n, hipMemcpyHostToDevice); //����� ���������
	//hipMemcpy(d_x0, h_x0, sizeof(double)*n, hipMemcpyHostToDevice); //��������� �����������
	//Kernel launch
	createMatrix << <blocksPerGrid, threadsPerBlock >> > (d_A, dx_int, n);
	create_F << <blocks, blockSize >> > (dx_int, f_d, n);
	cudaStat1 = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(f_h, f_d, sizeof(double)*n, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	cusolver_status = hipsolverDnDgeqrf_bufferSize(cusolverH, m, m, d_A, lda, &lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
	assert(hipSuccess == cudaStat1);

	cusolver_status = hipsolverDnDgeqrf(cusolverH, m, m, d_A, lda, d_tau, d_work, lwork, devInfo);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

	cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);

	printf("after geqrf: info_gpu = %d\n", info_gpu);
	assert(0 == info_gpu);


	cusolver_status = hipsolverDnDormqr(cusolverH, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T, m, nrhs, m, d_A, lda, d_tau, f_d, ldb, d_work, lwork, devInfo);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

	// check if QR is good or not
	cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);

	printf("after ormqr: info_gpu = %d\n", info_gpu);
	assert(0 == info_gpu);

	cublas_status = hipblasDtrsm(cublasH, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, nrhs, &one, d_A, lda, f_d, ldb);
	cudaStat1 = hipDeviceSynchronize();
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
	assert(hipSuccess == cudaStat1);

	cudaStat1 = hipMemcpy(xc, f_d, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);

	hipMemcpy(f_d, f_h, sizeof(double)*n, hipMemcpyHostToDevice);
	double err = 0, err_f = 0;
	double sum = 0;
	//printf("\t Matrix A: \n");
	for (i = 0; i < m; i++)
	{
		err_f += 2.0 / (3 + i) - f_h[i];
		for (j = 0; j < m; j++)
		{
			//calc << <blocks, blockSize >> > (x_d, n, sum_d, i, j);
			//hipMemcpy(&sum_h, sum_d, sizeof(double), hipMemcpyDeviceToHost);
			//h_A[j * m + i] = sum_h / n;
			err += (1.0 / (3 + i + j) - 1.0 / ((3 + i)*(3 + j))) - h_A[j * m + i];
			if(j!=i) sum += abs(h_A[j*m + i] / h_A[i*m + i]);
		//	printf("%f   ", h_A[j*m + i]);
			//sum_h = 0.0;
			//hipMemcpy(sum_d, &sum_h, sizeof(double), hipMemcpyHostToDevice);
			//hipMemcpy(x_d, x_h, sizeof(double)*n, hipMemcpyHostToDevice);
			//calc << <blocks, blockSize >> > (x_d, n, sum_d, i, j + 2);
			//hipMemcpy(&sum_h, sum_d, sizeof(double), hipMemcpyDeviceToHost);
		}
	//	printf("\n");
	}
	//printf("norm= %f\n", sum);
	//printf("\t Right part: \n");
	//for (i = 0; i < n; i++)
		//printf("%f  ", f_h[i]);
	printf("\t Error\n");
	//printf("error= %f\nerror_f= %f\n", err / n / n, err_f / n);
	printf("Error= %f\n", err_f / n);
	//hipEventRecord(start, 0);
	//while (eps > EPS)
	//{
	//	k++;
	//	hipMemcpy(d_delta, h_delta, sizeof(double)*n, hipMemcpyHostToDevice);
	//	//Solve << <blocks, blockSize >> > (d_A, f_d, d_x0, d_x1, n);
	//	KernelJacobi << <blocks, blockSize >> > (d_A, f_d, d_x0, d_x1, n);
	//	EpsJacobi << <blocks, blockSize >> > (d_x0, d_x1, d_delta, n);
	//	hipMemcpy(h_delta, d_delta, sizeof(double)*n, hipMemcpyDeviceToHost);
	//	eps = 0;
	//	for (j = 0; j < n; j++)
	//	{
	//		eps += h_delta[j];
	//		h_delta[j] = 0;
	//	}

	//	eps /= n;
	//	//printf("\n Eps[%d]=%f\n ", k, eps);
	//}
	//hipMemcpy(h_x1, d_x0, sizeof(double)*n, hipMemcpyDeviceToHost);
	for (i = 0; i < n; i++)
		//printf("x[%d]= %f\n", i, xc[i]);
	_resolution << <blocks, blockSize >> > (dx_int, f_d, x_d, n);
	hipMemcpy(x_h, x_d, sizeof(double)*n, hipMemcpyDeviceToHost);
	/*hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);*/
	for (i = 0; i < n; i++)
		printf("%f\n", x_h[i]);
	//printf("result=%f\n", sum_h / n);
	//hipFree(sum_d);
	hipFree(x_d);
	hipFree(d_A);
	hipFree(dx_int);
	if (d_tau) hipFree(d_tau);
	if (f_d) hipFree(f_d);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	//hipFree(d_delta);
	//hipFree(d_x0);
	//hipFree(d_x1);
	delete[] h_A;
	//delete[] x_h;
	delete[] f_h;
	//delete[] h_delta;
	//delete[] h_x0;
	//delete[] h_x1;
	if (cublasH) hipblasDestroy(cublasH);
	if (cusolverH) hipsolverDnDestroy(cusolverH);
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
